#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides

__device__ size_t compute_addr(size_t idx, CudaVec shape, CudaVec strides, size_t offset) {
  size_t dim_stride = 1;
  size_t addr = offset;

  for(int i = shape.size - 1; i >= 0; --i) {
    size_t dim_ind = size_t(idx / dim_stride);
    dim_ind = dim_ind % shape.data[i];
    dim_stride *= shape.data[i];
    addr += dim_ind * strides.data[i];
  }

  return addr;
}

__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact operation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size)
    out[gid] = a[compute_addr(gid, shape, strides, offset)];
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}



__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the element-wise set operation.  This should effectively map a single entry in the 
   * compact input a, to the corresponding item (at location gid) in the non-compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of a array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size)
    out[compute_addr(gid, shape, strides, offset)] = a[gid];
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
  // assert(false && "Not Implemented");
  /// END SOLUTION
}



__global__ void ScalarSetitemKernel(const scalar_t val, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the scalar set operation.  This should effectively map a single entry 
   * to the corresponding item (at location gid) in the non-compact array out.
   * 
   * Args:
   *   val: value to set to
   *   out: CUDA point to out array
   *   size: number of elements in out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size)
    out[compute_addr(gid, shape, strides, offset)] = val;
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
  // assert(false && "Not Implemented");
  /// END SOLUTION
}



////////////////////////////////////////////////////////////////////////////////
// Trignometric operations
////////////////////////////////////////////////////////////////////////////////
/*
CUDA:
__global__ void Sin/cos(float *angles, float *sineValues, float *cosineValues, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within the array bounds
    if (idx < numElements) {
        float angle = angles[idx];
        sineValues[idx] = sinf(angle);
        cosineValues[idx] = cosf(angle);
    }
} 
*/

__global__ void EwiseSinKernel(const scalar_t* a, scalar_t* out, size_t size) 
{
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) 
  {
    float angle = a[gid];
    out[gid] = sinf(angle);
  }
}

void EwiseSin(const CudaArray& a, CudaArray* out) {
  
  CudaDims dim = CudaOneDim(out->size);
  EwiseSinKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseCosKernel(const scalar_t* a, scalar_t* out, size_t size) 
{
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) 
  {
    float angle = a[gid];
    out[gid] = sinf(angle);
  }
}

void EwiseCos(const CudaArray& a, CudaArray* out) {
  
  CudaDims dim = CudaOneDim(out->size);
  EwiseCosKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}







////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * b[gid];
}

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Multiply together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * val;
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Multiply a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / b[gid];
}

void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Divide two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / val;
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Divide a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = pow(a[gid], val);
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Divide a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    scalar_t maximum = a[gid];

    if(b[gid] > maximum)
      maximum = b[gid];
    
    out[gid] = maximum;
  }
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Maximum of two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    scalar_t maximum = a[gid];

    if(val > maximum)
      maximum = val;
    
    out[gid] = maximum;
  }}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Maximum of a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseEqKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] == b[gid];
}

void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Equality of two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseEqKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarEqKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] == val;
}

void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Equality of a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarEqKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseGeKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] >= b[gid];
}

void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Greater than equal to of two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseGeKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarGeKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] >= val;
}

void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Greater or Equality of a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarGeKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseLogKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = log(a[gid]);
}

void EwiseLog(const CudaArray& a, CudaArray* out) {
  /**
   * Log of one CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseLogKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseExpKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = exp(a[gid]);
}

void EwiseExp(const CudaArray& a, CudaArray* out) {
  /**
   * Exp of one CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseExpKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseTanhKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = tanh(a[gid]);
}

void EwiseTanh(const CudaArray& a, CudaArray* out) {
  /**
   * Tanh of one CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseTanhKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////


__global__ void MatmulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, 
                            uint32_t M, uint32_t N, uint32_t P) {
  /// BEGIN SOLUTION
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;
  
  // if (i < M && j < P) {
  //   C[i][j] = 0;
  //   A[i][j] + B[i][j];
  
  // }
  __shared__ float a_tile[TILE][TILE];
  __shared__ float b_tile[TILE][TILE];
  float curr_val = 0;

  int row = blockIdx.y * TILE + threadIdx.y;
  int col = blockIdx.x * TILE + threadIdx.x;

  for(int i = 0; i < (N + TILE - 1) / TILE; ++i) {
    int a_col = i * TILE + threadIdx.x;
    a_tile[threadIdx.y][threadIdx.x] = 0;

    if(row < M && a_col < N)
      a_tile[threadIdx.y][threadIdx.x] = a[row * N + a_col];

    int b_row = i * TILE + threadIdx.y;
    b_tile[threadIdx.y][threadIdx.x] = 0;

    if(b_row < N && col < P)
      b_tile[threadIdx.y][threadIdx.x] = b[b_row * P + col];

    __syncthreads();

    for(int j = 0; j < TILE; ++j)
      curr_val += a_tile[threadIdx.y][j] * b_tile[j][threadIdx.x];

    __syncthreads();
  }

  if(row < M && col < P)
    out[row * P + col] = curr_val;

  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN SOLUTION
  dim3 grid = dim3((P + TILE - 1) / TILE, (M + TILE - 1) / TILE, 1);
  dim3 block = dim3(TILE, TILE, 1);
  MatmulKernel<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////


__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size) {
    scalar_t max;
    for(int i = 0; i < reduce_size; ++i) {
      if(!i)
        max = a[gid * reduce_size + i];

      else if(a[gid * reduce_size + i] > max)
        max = a[gid * reduce_size + i];

    }
    
    out[gid] = max;
  }
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
  // assert(false && "Not Implemented");
  /// END SOLUTION
}



__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size) {
    scalar_t sum = 0.0;
    for(int i = 0; i < reduce_size; ++i)
      sum += a[gid * reduce_size + i];
    
    out[gid] = sum;
  }
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
  // assert(false && "Not Implemented");
  /// END SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);


  m.def("EwiseSin", EwiseSin);
  m.def("EwiseCos", EwiseCos);

}
